#include <pybind11/pybind11.h>
#include <string>
#include <tuple>
#include <hip/hip_runtime.h>
#include <exception>
#include <sstream>

using namespace std;
namespace py = pybind11;

#define CONCAT(a, b) a##b
#define CONCAT3(a, b, c) a##b##c
#define STR(x) #x
#define GETTER(prop) \
    const auto CONCAT(get_, prop)() const { return this->prop; }
#define PROP(name, doc) def_property_readonly(STR(name), &CudaDeviceInfo::CONCAT(get_, name), doc)
#define MACRO_STRINGIFY(x) STR(x)

class CudaError : public exception
{
private:
    const char *reason;

public:
    CudaError(const char *error_str)
        : reason(error_str)
    {
    }
    const char *what() const noexcept override
    {
        return this->reason;
    }
};

class CudaDeviceInfo
{
private:
    int id;                            /* device id of device */
    string name;                       /* ASCII string identifying device */
    tuple<int, int> computeCapability; /* compute capability */
    size_t totalGlobalVmem;            /* Global memory available on device in bytes */
    tuple<int, int, int> pciId;        /* PCI bus ID, PCI device ID, PCI domain ID of this device */
    bool isTccDriver;                  /* true if device is a Tesla device using TCC driver, false otherwise  */

public:
    GETTER(id)
    GETTER(name)
    GETTER(computeCapability)
    GETTER(totalGlobalVmem)
    GETTER(pciId)
    GETTER(isTccDriver)

public:
    CudaDeviceInfo(int id)
    {
        hipDeviceProp_t prop = {0};
        hipError_t error = hipGetDeviceProperties(&prop, id);
        if (error != hipSuccess)
        {
            auto err_str = hipGetErrorString(error);
            throw CudaError(err_str);
        }
        this->id = id;
        this->name = prop.name;
        this->computeCapability = make_tuple(prop.major, prop.minor);
        this->totalGlobalVmem = prop.totalGlobalMem;
        this->pciId = make_tuple(prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
        this->isTccDriver = prop.tccDriver;
    }
};

string repr_for_info(const CudaDeviceInfo &a)
{
    auto [ccx, ccy] = a.get_computeCapability();
    auto [pcix, pciy, pciz] = a.get_pciId();
    string tcc = a.get_isTccDriver()
                     ? "True"
                     : "False";
    stringstream ss;
    ss << "{" << endl
       << "\t.Id = " << a.get_id() << endl
       << "\t.Name = " << a.get_name() << endl
       << "\t.Compute Capability = " << ccx << "." << ccy << endl
       << "\t.Total Global Video Memory = " << a.get_totalGlobalVmem() / 1024.0 / 1024.0 << "MBytes" << endl
       << "\t.PCI Id = " << pcix << "." << pciy << "." << pciz << endl
       << "\t Is using TCC driver = " << tcc << endl
       << "}" << endl;
    string output;
    ss >> output;
    return output;
}

int getCudaDeviceCount()
{
    int cnt = 0;
    hipError_t error = hipGetDeviceCount(&cnt);
    if (error != hipSuccess)
    {
        auto err_str = hipGetErrorString(error);
        throw CudaError(err_str);
    }
    return cnt;
}

PYBIND11_MODULE(cuda_available, m)
{
    m.doc() = R"pbdoc(
        cuda_available module
        -----------------------

        .. currentmodule:: cuda_available

        .. autosummary::
           :toctree: _generate

           CudaDeviceInfo
           getCudaDeviceCount
    )pbdoc";
    py::class_<CudaDeviceInfo>(m, "CudaDeviceInfo", R"pbdoc(
        Get device infomation of cuda device with given device_id
    )pbdoc")
        .def(py::init<int>())
        .PROP(id, "device id of device")
        .PROP(name, "ASCII string identifying device")
        .PROP(computeCapability, "compute capability")
        .PROP(totalGlobalVmem, "Global memory available on device in bytes")
        .PROP(pciId, "PCI bus ID, PCI device ID, PCI domain ID of this device")
        .PROP(isTccDriver, "true if device is a Tesla device using TCC driver, false otherwise")
        .def("__repr__", &repr_for_info)
        .def("__str__", &repr_for_info);

    m.def("getCudaDeviceCount", &getCudaDeviceCount, R"pbdoc(
        Get available cuda device count
    )pbdoc");
    py::register_local_exception<CudaError>(m, "CudaError");

#ifdef VERSION_INFO
    m.attr("__version__") = MACRO_STRINGIFY(VERSION_INFO);
#else
    m.attr("__version__") = "dev";
#endif
}